#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void add(int* a, int* b, int* c ){
    int i = threadIdx.x + blockDim.x* 
    c[i] = a[i] + b[i];
}

__managed__ int vector_a[256],vector_b[256],vector_c[256];

int main(int i = 0; i< 256; i++ ){

    vector_a[i] = i;

    vector_b[i] = 256 - i;

    add<<<1,256>>>(vector_a,vector_b , vector_c);

    hipDeviceSynchronize();
}






